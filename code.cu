/*
This program generates 2 N*N matrices and then multiplies them on a GPU
*/


#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<hip/hip_runtime.h>
//#define N 100

__global__ void multiply(float* A, float* B, float* C, int K)
{
	//printf("\n Entered kernel");
	int index1=blockIdx.x*blockDim.x+threadIdx.x;
	int index2=blockIdx.y*blockDim.y+threadIdx.y; 
	//printf("\n Index 1 = %d", index1);
	//printf("\n Index 2 = %d", index2);
	float sum=0.0;
	for (int i=0;i<K;i++)
	{
		sum+=A[index2*K+i]*B[i*K+index1];
	}
	
	C[index2*K+index1]=sum;
}

float** Make2DfloatArray(int arraySizeX, int arraySizeY) {
float** theArray;
theArray = (float**) malloc(arraySizeX*sizeof(float*));
int i;
for (i = 0; i < arraySizeX; i++)
   theArray[i] = (float*) malloc(arraySizeY*sizeof(float));
int j;

for (i=0;i<arraySizeX;i++)
{
    for (j=0;j<arraySizeY;j++)
    {
        theArray[i][j]=rand()%100;
    }
}

   return theArray;
}

void init_zeros(float** matrix, int K)
{
	int i,j;
	for (i=0;i<K;i++)
	{	
		for (j=0;j<K;j++)
		{
			matrix[i][j]=0;
		}
	}
}

float* Make1DfloatArray(int arraySizeX) {
float* theArray;
theArray = (float*)malloc(arraySizeX*sizeof(float));
int i;
for (i=0;i<arraySizeX;i++)
{
    theArray[i]=0.0;
}

   return theArray;
}

void printmat(float** matrix, int K)
{
	int i,j;
	
	for (i=0;i<K;i++)
	{	
		printf("\n");
		for (j=0;j<K;j++)
		{
			printf("%f \t",matrix[i][j]);
		}
	}
	printf("\n");
}

void printtofile(float** matrix, int K, char* filename)
{
	FILE *fp;
	fp=fopen(filename,"wt");
	int i,j;
	for (i=0;i<K;i++)
	{
		fprintf(fp, "\n");
		for (j=0;j<K;j++)
		{
			fprintf(fp, "%f\t", matrix[i][j]);
		}
	}
}

void printtofile(float* matrix, int K, char* filename)
{
	FILE *fp;
	fp=fopen(filename,"wt");
	int i,j;
	int counter=0;
	for (i=0;i<K;i++)
	{
		fprintf(fp, "\n");
		for (j=0;j<K;j++)
		{
			fprintf(fp, "%f\t", matrix[counter]);
			counter++;
		}
	}
}
	 
 
int main(int argc, char *argv[])
{
	const int K = 10000;
	const int blocks=K/200;
	const int threadblocks=K/blocks;
	float** M1=Make2DfloatArray(K,K);
	float** M2=Make2DfloatArray(K,K);
	float** Prod=Make2DfloatArray(K,K);
	
	init_zeros(Prod, K);
	
	float* M1_host_flat=Make1DfloatArray(K*K);
	float* M2_host_flat=Make1DfloatArray(K*K);
	float* Prod_host_flat=Make1DfloatArray(K*K);
	
	float* M1_device_flat;
	float* M2_device_flat;
	float* Prod_device_flat;
	int* K_device;
	
	printtofile(M1,K,"M1.txt");
	printtofile(M2,K,"M2.txt");
	printtofile(Prod,K,"Prod.txt");
	int counter=0;
	int i,j;
	for(i=0;i<K;i++)
	{
		for(j=0;j<K;j++)
		{
			M1_host_flat[counter]=M1[i][j];
			M2_host_flat[counter]=M2[i][j];
			Prod_host_flat[counter]=Prod[i][j];
			counter+=1;
			
		}
	}
	
	hipMalloc((void **) &M1_device_flat, sizeof(float)*K*K);
	hipMalloc((void **) &M2_device_flat, sizeof(float)*K*K);
	hipMalloc((void **) &Prod_device_flat, sizeof(float)*K*K);
	hipMalloc((void **) &K_device, sizeof(int));
	
	hipMemcpy(M1_device_flat, M1_host_flat, sizeof(float)*K*K, hipMemcpyHostToDevice);
	hipMemcpy(M2_device_flat, M2_host_flat, sizeof(float)*K*K, hipMemcpyHostToDevice);
	hipMemcpy(Prod_device_flat, Prod_host_flat, sizeof(float)*K*K, hipMemcpyHostToDevice);
	hipMemcpy(K_device, &K, sizeof(int), hipMemcpyHostToDevice);
	//Kernel call
	dim3 threads(threadblocks,threadblocks);
	dim3 grid(blocks,blocks);
	printf("\n Calling the multiply kernel");
	multiply<<<grid,threads>>>(M1_device_flat,M2_device_flat,Prod_device_flat, K); 
	//Copy data back to host
	printf("\n Back in host\n");
	hipMemcpy(Prod_host_flat, Prod_device_flat, sizeof(int)*K*K, hipMemcpyDeviceToHost);	
	/*
	counter=0;
	printf("\n");
	printf("\n");
	printf("\n");
	for (i=0;i<K;i++)
	{
		//fprintf(results_file,"\n");
		printf("\n");
		for (j=0;j<K;j++)
		{
			printf("%f ", Prod_host_flat[counter]);
			counter+=1;
		}
	}
	printf("\n");
	*/
	printtofile(Prod_host_flat,K,"Prod_result.txt");
	
	
	return 0;	
}
