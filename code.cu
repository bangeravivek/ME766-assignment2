/*
This program generates 2 N*N matrices and then multiplies them on a GPU
*/


#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<hip/hip_runtime.h>
#include<unistd.h>
//#define N 100

__global__ void multiply(float* A, float* B, float* C, int K)
{
	/*
	The Kernel is a 2D grid. Tried doing the same with a 1D grid but it requires 2 for loops
	*/
	//printf("\n Entered kernel");
	int index1=blockIdx.x*blockDim.x+threadIdx.x;
	int index2=blockIdx.y*blockDim.y+threadIdx.y; 
	float sum=0.0;
	for (int i=0;i<K;i++)
	{
		sum+=A[index2*K+i]*B[i*K+index1];
	}
	
	C[index2*K+index1]=sum;
}

float** Make2DfloatArray(int arraySizeX, int arraySizeY) {
float** theArray;
theArray = (float**) malloc(arraySizeX*sizeof(float*));
int i;
for (i = 0; i < arraySizeX; i++)
   theArray[i] = (float*) malloc(arraySizeY*sizeof(float));
int j;

for (i=0;i<arraySizeX;i++)
{
    for (j=0;j<arraySizeY;j++)
    {
        theArray[i][j]=rand()%5;
    }
}

   return theArray;
}

void init_zeros(float** matrix, int K)
{
	int i,j;
	for (i=0;i<K;i++)
	{	
		for (j=0;j<K;j++)
		{
			matrix[i][j]=0;
		}
	}
}

float* Make1DfloatArray(int arraySizeX) {
float* theArray;
theArray = (float*)malloc(arraySizeX*sizeof(float));
int i;
for (i=0;i<arraySizeX;i++)
{
    theArray[i]=0.0;
}

   return theArray;
}

void printmat(float** matrix, int K)
{
	int i,j;
	
	for (i=0;i<K;i++)
	{	
		printf("\n");
		for (j=0;j<K;j++)
		{
			printf("%f \t",matrix[i][j]);
		}
	}
	printf("\n");
}

void printtofile(float** matrix, int K, char* filename)
{
	FILE *fp;
	fp=fopen(filename,"wt");
	int i,j;
	for (i=0;i<K;i++)
	{
		fprintf(fp, "\n");
		for (j=0;j<K;j++)
		{
			fprintf(fp, "%f\t", matrix[i][j]);
		}
	}
}

void printtofile1D(float* matrix, int K, char* filename)
{
	FILE *fp;
	fp=fopen(filename,"wt");
	int i,j;
	int counters=0;
	for (i=0;i<K;i++)
	{
		fprintf(fp, "\n");
		for (j=0;j<K;j++)
		{
			fprintf(fp, "%f \t", matrix[counters]);
			counters++;
		}
	}
}

void freese(int sizeX, float** ptr)
{
    int i;
     for (i=0;i<sizeX;i++)
        free(ptr[i]);
    free(ptr);
}

	 
 
int main(int argc, char *argv[])
{

	const int K = 1000;
	const int blocks=K/20;
	const int threadblocks=K/blocks;
	float** M1=Make2DfloatArray(K,K);
	float** M2=Make2DfloatArray(K,K);
	float** Prod=Make2DfloatArray(K,K);
	
	init_zeros(Prod, K);
	
	float* M1_host_flat=Make1DfloatArray(K*K);
	float* M2_host_flat=Make1DfloatArray(K*K);
	float* Prod_host_flat=Make1DfloatArray(K*K);
	
	float* M1_device_flat;
	float* M2_device_flat;
	float* Prod_device_flat;
	int* K_device;
	printf("\n Everything initialized");

	printtofile(M1,K,"M1.txt");
	printtofile(M2,K,"M2.txt");
	printtofile(Prod,K,"Prod.txt");

	printtofile(M1,K,"M1.txt");
	printtofile(M2,K,"M2.txt");
	printtofile(Prod,K,"Prod.txt");

	int counter=0;
	int i,j;
	for(i=0;i<K;i++)
	{
		for(j=0;j<K;j++)
		{
			M1_host_flat[counter]=M1[i][j];
			M2_host_flat[counter]=M2[i][j];
			Prod_host_flat[counter]=Prod[i][j];
			counter+=1;
			
		}
	}
	
	printf("\n Converted to flat");
	hipMalloc((void **) &M1_device_flat, sizeof(float)*K*K);
	hipMalloc((void **) &M2_device_flat, sizeof(float)*K*K);
	hipMalloc((void **) &Prod_device_flat, sizeof(float)*K*K);
	hipMalloc((void **) &K_device, sizeof(int));
	
	hipMemcpy(M1_device_flat, M1_host_flat, sizeof(float)*K*K, hipMemcpyHostToDevice);
	hipMemcpy(M2_device_flat, M2_host_flat, sizeof(float)*K*K, hipMemcpyHostToDevice);
	hipMemcpy(Prod_device_flat, Prod_host_flat, sizeof(float)*K*K, hipMemcpyHostToDevice);
	hipMemcpy(K_device, &K, sizeof(int), hipMemcpyHostToDevice);
	//Kernel call
	dim3 threads(threadblocks,threadblocks);
	dim3 grid(blocks,blocks);
	printf("\n Calling the multiply kernel");
	multiply<<<grid,threads>>>(M1_device_flat,M2_device_flat,Prod_device_flat, K); 
	//Copy data back to host
	printf("\n Back in host\n");
	hipMemcpy(Prod_host_flat, Prod_device_flat, sizeof(int)*K*K, hipMemcpyDeviceToHost);	
	/*
	counter=0;
	printf("\n");
	printf("\n");
	printf("\n");
	for (i=0;i<K;i++)
	{
		//fprintf(results_file,"\n");
		printf("\n");
		for (j=0;j<K;j++)
		{
			printf("%f ", Prod_host_flat[counter]);
			counter+=1;
		}
	}
	printf("\n");
	*/
	printtofile1D(Prod_host_flat,K,"Prod_result.txt");
	sleep(5);
	hipFree(M1_device_flat);
	hipFree(M2_device_flat);
	hipFree(Prod_device_flat);
	hipFree(K_device);
	freese(K,M1);
	freese(K,M2);
	freese(K,Prod);
	free(M1_host_flat);
	free(M2_host_flat);
	free(Prod_host_flat);
	
	return 0;	
}
