/*
This program generates 2 N*N matrices and then multiplies them on a GPU
*/


#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<hip/hip_runtime.h>
#include<unistd.h>
//#define N 100

__global__ void multiply(float* A, float* B, float* C, int K)
{
	/*
	The Kernel is a 2D grid. Tried doing the same with a 1D grid but it requires 2 for loops
	*/
	//printf("\n Entered kernel");
	int index1=blockIdx.x*blockDim.x+threadIdx.x;
	int index2=blockIdx.y*blockDim.y+threadIdx.y; 
	float sum=0.0;
	for (int i=0;i<K;i++)
	{
		sum+=A[index2*K+i]*B[i*K+index1];
	}
	
	C[index2*K+index1]=sum;
}

float** Make2DfloatArray(int arraySizeX, int arraySizeY) {
float** theArray;
theArray = (float**) malloc(arraySizeX*sizeof(float*));
int i;
for (i = 0; i < arraySizeX; i++)
   theArray[i] = (float*) malloc(arraySizeY*sizeof(float));
int j;

for (i=0;i<arraySizeX;i++)
{
    for (j=0;j<arraySizeY;j++)
    {
        theArray[i][j]=rand()%5;
    }
}

   return theArray;
}

void init_zeros(float** matrix, int K)
{
	int i,j;
	for (i=0;i<K;i++)
	{	
		for (j=0;j<K;j++)
		{
			matrix[i][j]=0;
		}
	}
}

float* Make1DfloatArray(int arraySizeX) {
float* theArray;
theArray = (float*)malloc(arraySizeX*sizeof(float));
int i;
for (i=0;i<arraySizeX;i++)
{
    theArray[i]=0.0;
}

   return theArray;
}

void printmat(float** matrix, int K)
{
	int i,j;
	
	for (i=0;i<K;i++)
	{	
		printf("\n");
		for (j=0;j<K;j++)
		{
			printf("%f \t",matrix[i][j]);
		}
	}
	printf("\n");
}

void printtofile(float** matrix, int K, char* filename)
{
	FILE *fp;
	fp=fopen(filename,"wt");
	int i,j;
	for (i=0;i<K;i++)
	{
		fprintf(fp, "\n");
		for (j=0;j<K;j++)
		{
			fprintf(fp, "%f\t", matrix[i][j]);
		}
	}
}

void printtofile1D(float* matrix, int K, char* filename)
{
	FILE *fp;
	fp=fopen(filename,"wt");
	int i,j;
	int counters=0;
	for (i=0;i<K;i++)
	{
		fprintf(fp, "\n");
		for (j=0;j<K;j++)
		{
			fprintf(fp, "%f \t", matrix[counters]);
			counters++;
		}
	}
}

void freese(int sizeX, float** ptr)
{
    int i;
     for (i=0;i<sizeX;i++)
        free(ptr[i]);
    free(ptr);
}

	 
 
int main(int argc, char *argv[])
{

	const int K = 100;
	const int blocks=K/20;
	const int threadblocks=K/blocks;
	float** M1=Make2DfloatArray(K,K);
	float** M2=Make2DfloatArray(K,K);
	float** Prod=Make2DfloatArray(K,K);
	
	hipEvent_t start, stop, start_kernel, stop_kernel;
	float time, time_kernel;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventCreate(&start_kernel);
	hipEventCreate(&stop_kernel);
		
	init_zeros(Prod, K);
	
	float* M1_host_flat=Make1DfloatArray(K*K);
	float* M2_host_flat=Make1DfloatArray(K*K);
	float* Prod_host_flat=Make1DfloatArray(K*K);
	
	float* M1_device_flat;
	float* M2_device_flat;
	float* Prod_device_flat;
	int* K_device;
	printf("\n Everything initialized");

	printtofile(M1,K,"M1.txt");
	printtofile(M2,K,"M2.txt");
	printtofile(Prod,K,"Prod.txt");


	int counter=0;
	int i,j;
	for(i=0;i<K;i++)
	{
		for(j=0;j<K;j++)
		{
			M1_host_flat[counter]=M1[i][j];
			M2_host_flat[counter]=M2[i][j];
			Prod_host_flat[counter]=Prod[i][j];
			counter+=1;
			
		}
	}
	
	//printf("\n Converted to flat");
	hipEventRecord(start,0);
	hipMalloc((void **) &M1_device_flat, sizeof(float)*K*K);
	hipMalloc((void **) &M2_device_flat, sizeof(float)*K*K);
	hipMalloc((void **) &Prod_device_flat, sizeof(float)*K*K);
	hipMalloc((void **) &K_device, sizeof(int));
	
	hipMemcpy(M1_device_flat, M1_host_flat, sizeof(float)*K*K, hipMemcpyHostToDevice);
	hipMemcpy(M2_device_flat, M2_host_flat, sizeof(float)*K*K, hipMemcpyHostToDevice);
	hipMemcpy(Prod_device_flat, Prod_host_flat, sizeof(float)*K*K, hipMemcpyHostToDevice);
	hipMemcpy(K_device, &K, sizeof(int), hipMemcpyHostToDevice);
	//Kernel call
	dim3 threads(threadblocks,threadblocks);
	dim3 grid(blocks,blocks);
	hipEventRecord(start_kernel,0);
	//printf("\n Calling the multiply kernel");
	multiply<<<grid,threads>>>(M1_device_flat,M2_device_flat,Prod_device_flat, K); 
	hipEventRecord(stop_kernel,0);
	//Copy data back to host
	//printf("\n Back in host\n");
	hipMemcpy(Prod_host_flat, Prod_device_flat, sizeof(int)*K*K, hipMemcpyDeviceToHost);	
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&time, start, stop);
	hipEventElapsedTime(&time_kernel, start_kernel, stop_kernel);
	printf("\nTime for kernel with data transfer = %f ms \n", time);
	printf("\nTime for kernel without data transfer = %f ms \n", time_kernel); 
	/*
	counter=0;
	printf("\n");
	printf("\n");
	printf("\n");
	for (i=0;i<K;i++)
	{
		//fprintf(results_file,"\n");
		printf("\n");
		for (j=0;j<K;j++)
		{
			printf("%f ", Prod_host_flat[counter]);
			counter+=1;
		}
	}
	printf("\n");
	*/
	
	printtofile1D(Prod_host_flat,K,"Prod_result.txt");
	
	hipFree(M1_device_flat);
	hipFree(M2_device_flat);
	hipFree(Prod_device_flat);
	hipFree(K_device);
	freese(K,M1);
	freese(K,M2);
	freese(K,Prod);
	free(M1_host_flat);
	free(M2_host_flat);
	free(Prod_host_flat);
	
	return 0;	
}
